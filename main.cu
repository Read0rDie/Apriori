
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

long int factorial(int x);
long int nCr(int n, int r);

struct timeval start, end;

void starttime(){
	gettimeofday(&start, 0);
}

void endtime(const char * c){
	gettimeofday(&end, 0);
	double elapsed = (end.tv_sec - start.tv_sec) * 1000.0 + (end.tv_usec - start.tv_usec)/ 1000.0;
	printf("%s: %f ms\n", c, elapsed);
}

int main() {
	
	//Variable declarations
	FILE *fPointer;
//	char singleLine[100];
	int max = 0; //Contains the largest integer occurence in the given database
	int size = 0; //Contains the number of lines in the given database
	int cardinality = 1; //Contains the initial cardinality of the item sets
    	int temp;
	int i = 0;
	int j, k, num, count;
	int mSupport = 8000; //Contains the support count; set to approx 10% of all transactions
	char val;
//	int THREADS_PER_BLOCK = 1024;

	//While loop that traverses through the database and returns the number of transactions  
	fPointer = fopen("retail.dat", "r"); 
   	fscanf(fPointer, "%c", &val);
   	while(!feof(fPointer)){
        	if(val == '\n'){
            		size++;
        	}
       		fscanf(fPointer, "%c", &val);
    	}
    	fclose(fPointer);
  //  	printf("\nNumber of Transcations: %d\n", size);

    	fPointer = fopen("retail.dat", "r");
   	fscanf(fPointer, "%d", &temp);
 //	printf("ID number of first item: %d\n", temp);
	
	//Traverses through each transaction in order to find the max value.
    	while(!feof(fPointer)){
        	fscanf(fPointer, "%d", &temp);
        	if(max < temp){
            		max = temp;
        	}
    	}	
   	fclose(fPointer);


printf("DATA FILE PARSED\n");
printf("=========================================\n");
printf("Total number of transactions found: %d\n", size);
printf("Total number of unique items: %d\n", max + 1);
printf("=========================================\n");
printf("APRIORI IMPLEMENTATION BEGINS\n");
  // 	printf("Largest ID number found: %d\n", max);

	//Counting the number of necessary blocks.
	//int numblocks = size/THREADS_PER_BLOCK;
	//if (size % THREADS_PER_BLOCK != 0) {
	//	numblocks++; 
	//}
    	
//	printf("\nSH: initializing transaction array\n");
starttime();
	//Creation of table
	char *cTable = (char*)malloc(sizeof(char) * (max + 1) * size); //Allocates an array of characters for each transaction	
	
	for(i=0; i < (max+1)*size; i++) {
	//	memset(cTable[i], '\0', sizeof(char) * (max + 1) * size); //Initialize all values to 0.
		cTable[i] = '\0';
	}



//	printf("SH: initialization of transaction array COMPLETE\n");
	
//	printf("\nSH: populating transaction array\n");
		
    	char line[400];
    	char *cNum;
    	fPointer = fopen("retail.dat", "r");
	for(i = 0; i < size; i++){
		fgets(line, 400, fPointer);

        	cNum = strtok(line, " \n");
        	
		while(cNum != NULL){
            		num = atoi(cNum);
            		cTable[i * (max + 1) + num] = '1';
            		cNum = strtok(NULL, " \n");
        	}	
    	}

//	printf("SH: populating transaction array COMPLETE\n");

//	printf("\nSH: initializing cardinality '1' sets\n");
	
	//Creates a frequency table of iem sets with a Cardinality of 1; where the array index represents the item 
	//number. All the items have their counts initially set to zero
	int fTable[max + 1];
	for(i = 0; i < max + 1; i++){
		fTable[i] = 0;
	}

//	printf("SH: initialization of cardinality '1' sets COMPLETE\n");

//	printf("\nSH: counting of cardinality '1' sets\n");

	//Iterate though the frequency table and count the occurences of each item in the transcations from the cTable
	for(i = 0; i < size; i++){
		for(j = 0; j < max + 1; j++){
			if(cTable[i * (max + 1) + j] == '1'){
				fTable[j]++;
			}
		}
	}

//	printf("SH: counting of cardinality '1' sets COMPLETE\n");

	//Alter the value of 'i' to check the counts of any particular item in the transcation data set
	i = 32;
	char sInt[6];
	snprintf(sInt, 6, "%d", i);

	//printf("\nThere are %d instances of item'%s'\n", fTable[i], sInt);

//	printf("\nSH: removing item sets whose counts are below the support threshold\n");

	//invalidating elements that are below the support count and counting the remaining eligible elements
	count = 0;
	for(i = 0; i < (max + 1); i++){
		if (fTable[i] < mSupport){
			fTable[i] = 0;
		}
		else{
			count++;
		}
	}

//	printf("SH: removal of item sets COMPLETE\n");
//	printf("\nRemaining items sets: %d\n", count);

	//creating new table consisting of only valid items
        int iTable[count];
        j = 0;
        for(i = 0; i < (max + 1); i++){
                if (fTable[i] != 0){
                        iTable[j] = i;			
                        j++;
                }
        }

	//creating a tabel to hold the current valid items item and their the a variable for the count of the count
	int * vTable = iTable;
	int lastCount = count;

	while(count > 1){
		cardinality++;
//		printf("\nSH: initializating new cardinality '%d' sets\n", cardinality);	
	
		//temporary array that will hold the new item sets		
		int temp[nCr(count, cardinality) * (cardinality + 1)];
		
//		printf("SH: initialization of new cardinality '%d' sets COMPLETE\n", cardinality);

//		printf("\nSH: initializating old cardinality '%d' sets\n", cardinality - 1);		

		//array of previous items sets
		int oldSets[nCr(lastCount, cardinality - 1) * cardinality];

		//array that hold one old item set for insertion into table
		int oldEntry[cardinality - 1];

//		printf("SH: initialization of old cardinality '%d' sets COMPLETE\n", cardinality - 1);
		
//		 printf("\nSH: populating old  cardinality '%d' sets\n", cardinality - 1); 

                //function populates old  item set
                k = 0;
                if(cardinality - 1 <= lastCount){
                        for(i = 0; (oldEntry[i] = i) < cardinality - 2; i++); 
                        for(i = 0; i < cardinality - 1; i++){
                                oldSets[(k * cardinality) + i] = vTable[oldEntry[i]];
                        }
                        k++;
                        for(;;){
                                for( i = cardinality - 2; i >= 0 && oldEntry[i] == (lastCount - (cardinality - 1) + i); i--);
                                if(i < 0){
                                        break;
                                }
                                else{
                                        oldEntry[i]++;
                                        for(++i; i < cardinality - 1; i++){
                                                oldEntry[i] = oldEntry[i - 1] + 1;
                                        }
                                        for(j = 0; j < cardinality - 1; j++){
                                                oldSets[(k * cardinality) + j] = vTable[oldEntry[j]];
                                        }
                                        k++;
                                }
                        }
                }

                for(i = 0; i < nCr(lastCount, cardinality - 1); i++){
                        oldSets[(i * cardinality) + cardinality - 1] = 0;
                }

//		printf("SH: populating of old cardinality '%d' sets COMPLETE\n", cardinality - 1);

		//array that will hold the information for a single item set before it is added to the 
		//array of all item sets
		int entry[cardinality];

//		printf("\nSH: populating cardinality '%d' sets\n", cardinality);

		//function populates new item set
		k = 0;
		if(cardinality <= count){
			for(i = 0; (entry[i] = i) < cardinality - 1; i++);			
			for(i = 0; i < cardinality; i++){
				temp[(k*(cardinality + 1)) + i] = vTable[entry[i]];
			}
			k++;
			for(;;){
				for( i = cardinality - 1; i >= 0 && entry[i] == (count - cardinality + i); i--);
				if(i < 0){
					break;
				}
				else{
					entry[i]++;
					for(++i; i < cardinality; i++){
						entry[i] = entry[i - 1] + 1;
					}
					for(j = 0; j < cardinality; j++){
						temp[(k*(cardinality + 1)) + j] = vTable[entry[j]];
					}
					k++;
				}
			}
		}


						      
		for(i = 0; i < nCr(count, cardinality); i++){
			temp[(i*(cardinality + 1)) + cardinality ] = 0;
		}

//		printf("SH: populating of cardinality '%d' sets COMPLETE\n", cardinality);

//		printf("\nSH: counting  cardinality '%d' sets\n", cardinality);
		
		//counting the amount of instances of the item sets amongst the transactions
		
		int found = 0; 
		int b = 0; 
		for(i = 0; i < size; i++){
			for(j = 0; j < nCr(count, cardinality); j++){
				found = 1;
				for(k = 0; k < cardinality; k++){
					b = temp[(j*(cardinality+1))+k];
					if(cTable[(i*(max+1))+b] != '1'){
						found = 0;
					}
				}
				if(found == 1){
					temp[(j*(cardinality + 1))+cardinality]++;
				}
			}
		}

//		printf("SH: counting of cardinality '%d' sets COMPLETE\n\n", cardinality);

//		printf("\nSH: counting old cardinality '%d' sets\n", cardinality - 1);

                //counting the amount of instances of the item sets amongst the transactions

                for(i = 0; i < size; i++){
                        for(j = 0; j < nCr(lastCount, cardinality - 1); j++){
                                int found = 1;
                                for(k = 0; k < cardinality -1; k++){
					int b = oldSets[(j*cardinality)+k];
                                        if(cTable[(i*(max + 1))+b] != '1'){
                                                found = 0;
                                        }
                                }
                                if(found == 1){
                                        oldSets[(j*cardinality) + cardinality - 1]++;
              			}                  
                        }
                }

//		printf("SH: counting of old cardinality '%d' sets COMPLETE\n\n", cardinality - 1);
/*
		for(i = 0; i <= cardinality; i++){
			if(i == cardinality){
				printf("Count\n");
			}
			else{
				printf("Item '%d'\t", (i+1));
			}
		}
		for(i = 0; i < nCr(count, cardinality); i ++){
                        for(j = 0; j <= cardinality; j++){
                                printf("%d\t\t", temp[(i*(cardinality + 1))+j]);
                        }
                         printf("\n");
                }
*/
//		printf("\nSH: removing item sets whose counts are below the support threshold\n");
		//invalidating elements that are below the support count and counting the remaining eligible elements
        	int tCount = count;
		lastCount = tCount;
		count = 0;
        	for(i = 0; i < nCr(tCount, cardinality); i++){
                	if (temp[(i*(cardinality + 1)) + cardinality] < mSupport){
                        	temp[(i * (cardinality + 1)) + cardinality] = 0;
                	}	
                	else{
                        	count++;
                	}
        	}		

  //      	printf("SH: removal of item sets COMPLETE\n");
    //    	printf("\nRemaining items sets: %d\n", count);

		//set Table of valid items
		char valid[max + 1];
		for(i = 0; i <= max; i++){
			valid[i] = '\0';
		}
		for(i = 0; i < nCr(tCount, cardinality); i++){
			for(j = 0; j < cardinality; j++){
				if(temp[(i * (cardinality + 1)) + cardinality] > 0){
					valid[temp[(i * (cardinality + 1)) + j]] = '1';
				}
			}
		}

        	//creating new table consisting of only valid items
        	int rTable[count];
		count = 0;
        	j = 0;
        	for(i = 0; i <= max; i++){
                	if (valid[i] == '1'){
                        	rTable[j] = i;
                        	j++;
				count++;
	                }
        	}	
		vTable = rTable;

		if(count == 0){
			printf("\n----Most Frequent Item Sets----\n\n");
	   
	        	for(i = 0; i < nCr(lastCount, cardinality - 1); i++){
				if(oldSets[(i * cardinality) + (cardinality-1)] > mSupport){
                                        printf("Set: {");
                                }
               			for(j = 0; j < cardinality; j++){
					if(oldSets[(i * cardinality) + (cardinality-1)] > mSupport){
                               			if(j == cardinality - 1){
							printf("}\t\tCount: %d\n", oldSets[(i * cardinality) + j]);
						}
						else{
							printf("'%d'", oldSets[(i * cardinality) + j]);
						}
                       		 	}	
               		 	}        
			}
			printf("\n");	
		}
	}

	endtime("Total parallelized Implementation Time");
}

//factorial function
long int factorial(int x){
	int count = x;
	while (count > 1){
		x = x * (count - 1);
		count--;
	}
	if(x == 0){
		x = 1;
	}	
	return x;	
}

//combinatorics function
long int nCr(int n, int r){
	int y;
	int z;
	int w = n - 1;
	int init = n;
	int x;
	if(r > (n-r)){
	y = r;	
	}
	else{
		y = (n-r);
	}

	z = n - y;
	while(z > 1){
		n = n * w;
		w--;
		z--;
	}
	if( r > (init - r)){
		x = n/factorial(init - r);
	}
	else{
		x = n/factorial(r);
	}
	
	return  x;

}
